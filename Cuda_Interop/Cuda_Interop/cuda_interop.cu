#include "hip/hip_runtime.h"
﻿#include "cuda_interop.h"
#include "hip/hip_runtime_api.h"
#include <iostream>
#include "nvcomp/cascaded.h"

void GraphicsResource::registerTexture()
{
    if (!isRegistered)
    {
        CHECK_ERROR(hipGraphicsGLRegisterImage(&resource, id, GL_TEXTURE_2D, cudaGraphicsMapFlagsNone), __FILE__, __LINE__);
        hipStreamCreate(&stream);
        hipDeviceSynchronize();
        isRegistered = true;
    }
}

void GraphicsResource::mapResource()
{
    if (!isMapped)
    {
//        count++;
//        if (!log_file.is_open())
//            log_file.open("debug_log.txt");
//        log_file << count << std::endl;
        CHECK_ERROR(hipGraphicsMapResources(1, &resource, stream), __FILE__, __LINE__);
        CHECK_ERROR(hipStreamSynchronize(stream), __FILE__, __LINE__);
        isMapped = true;
    }
}

void GraphicsResource::copyCudaArray()
{
    if (data_pointer == nullptr)
    {
        CHECK_ERROR(hipMalloc(&data_pointer, data_length), __FILE__, __LINE__);
    }
	CHECK_ERROR(hipGraphicsSubResourceGetMappedArray(&array, resource, 0, 0), __FILE__, __LINE__);
    CHECK_ERROR(hipMemcpy2DFromArray(data_pointer, width * sizeof(uchar4), array, 0, 0, width * sizeof(uchar4), height, hipMemcpyDeviceToDevice), __FILE__, __LINE__);
    //Debug
	//output_for_debug();
}

void GraphicsResource::unmapResource()
{
    if (!isRegistered || !isMapped)
        return;
	CHECK_ERROR(hipGraphicsUnmapResources(1, &resource, stream), __FILE__, __LINE__);
    CHECK_ERROR(hipStreamSynchronize(stream), __FILE__, __LINE__);
    isMapped = false;
}

void GraphicsResource::unregisterResource()
{
    if (!isRegistered)
        return;
	CHECK_ERROR(hipGraphicsUnregisterResource(resource), __FILE__, __LINE__);
    isRegistered = false;
}

void GraphicsResource::output_for_debug()
{
	if (!isFirstDebug)
		return;
	isFirstDebug = false;
	std::ofstream file;
	file.open("debug.ppm");
	void* test = malloc(data_length);
	CHECK_ERROR(hipMemcpy(test, data_pointer, data_length, hipMemcpyDeviceToHost), __FILE__, __LINE__);
	file << "P3" << std::endl
		<< "1920 1080" << std::endl
		<< "255" << std::endl;
	int texture_size = width * height;
	for (int i = 0; i < texture_size; i++)
	{
		unsigned char* c = (unsigned char*)test + i * 4;
		file << (int)*c << ' '
			<< (int)*(c + 1) << ' '
			<< (int)*(c + 2) << std::endl;
	}
	free(test);
	file.close();
}

//void GraphicsResource::compress()
//{
//    if (!isMapped)
//        return;
//    if (!isFirstCompress)
//    {
//        return;
//    }
//    isFirstCompress = false;
//    size_t* host_uncompressed_bytes;
//    const size_t chunk_size = 4096;
//    const size_t batch_size = (data_length + chunk_size -1) / chunk_size;
//    CHECK_ERROR(hipHostMalloc(&host_uncompressed_bytes, sizeof(size_t) * batch_size), __FILE__, __LINE__);
//    for (int i = 0; i < batch_size; i++)
//    {
//        if (i+1 < batch_size)
//        {
//            host_uncompressed_bytes[i] = chunk_size;
//        }
//        else
//        {
//            host_uncompressed_bytes[i] = data_length - (chunk_size*i);
//        }
//    }
//    void** host_uncompressed_ptrs;
//    CHECK_ERROR(hipHostMalloc(&host_uncompressed_ptrs, sizeof(size_t) * batch_size), __FILE__, __LINE__);
//    for (int chunk_index = 0; chunk_index < batch_size; chunk_index++)
//    {
//        host_uncompressed_ptrs[chunk_index] = (char*)data_pointer + chunk_index * chunk_size;
//    }
//    if (uncompressed_ptrs == nullptr)
//    {
//        hipMalloc(&uncompressed_bytes, sizeof(size_t) * batch_size);
//    }
//    if (uncompressed_bytes == nullptr)
//    {
//        hipMalloc(&uncompressed_ptrs, sizeof(size_t) * batch_size);
//    }
//    hipMemcpyAsync(uncompressed_bytes, host_uncompressed_bytes, sizeof(size_t) * batch_size, hipMemcpyHostToDevice, stream);
//    hipMemcpyAsync(uncompressed_ptrs, host_uncompressed_ptrs, sizeof(size_t) * batch_size, hipMemcpyHostToDevice, stream);
//    //free ptr
//    hipFree(host_uncompressed_bytes);
//    hipFree(host_uncompressed_ptrs);
//
//    CHECK_NVCOMP(nvcompBatchedCascadedCompressGetTempSize(batch_size, chunk_size, nvcompBatchedCascadedDefaultOpts, &temp_bytes), __FILE__, __LINE__);
//    CHECK_ERROR(hipMalloc(&temp_ptr, temp_bytes), __FILE__, __LINE__);
//    size_t max_out_bytes;
//    CHECK_NVCOMP(nvcompBatchedCascadedCompressGetMaxOutputChunkSize(chunk_size, nvcompBatchedCascadedDefaultOpts, &max_out_bytes), __FILE__, __LINE__);
//    void ** host_compressed_ptrs;
//    hipHostMalloc(&host_compressed_ptrs, sizeof(size_t) * batch_size);
//    for (int chunk_index = 0; chunk_index < batch_size; chunk_index++)
//    {
//        hipMalloc(&host_compressed_ptrs[chunk_index], max_out_bytes);
//    }
//    if (device_compressed_ptrs == nullptr)
//    {
//        hipMalloc(&device_compressed_ptrs, sizeof(size_t) * batch_size);
//    }
//    hipMemcpyAsync(device_compressed_ptrs, host_compressed_ptrs,
//            sizeof(size_t) * batch_size,hipMemcpyHostToDevice, stream);
//
//    //CHECK_ERROR(hipStreamSynchronize(stream), __FILE__, __LINE__);
//    if (device_compressed_bytes == nullptr)
//    {
//        hipMalloc(&device_compressed_bytes, sizeof(size_t) * batch_size);
//    }
//    CHECK_NVCOMP(nvcompBatchedCascadedCompressAsync(uncompressed_ptrs,
//                                                    uncompressed_bytes,
//                                                    chunk_size,
//                                                    batch_size,
//                                                    temp_ptr,
//                                                    temp_bytes,
//                                                    device_compressed_ptrs,
//                                                    device_compressed_bytes,
//                                                    nvcompBatchedCascadedDefaultOpts,
//                                                    stream), __FILE__, __LINE__);
//    hipStreamSynchronize(stream);
//    hipFree(temp_ptr);
//    hipFree(data_pointer);
//    hipFree(uncompressed_ptrs);
//    hipFree(uncompressed_bytes);


    //debug start
//    size_t* host_out_bytes;
//    hipHostMalloc(&host_out_bytes, sizeof(size_t)*batch_size);
//    hipMemcpy(host_out_bytes, device_compressed_bytes, sizeof(size_t) * batch_size, hipMemcpyDeviceToHost);
//    for (int i = 0; i < batch_size; i++)
//    {
//        log_file << "compressed bytes: " << *(host_out_bytes + i) << std::endl;
//    }
    //debug end

    //decompress
//    nvcompBatchedCascadedGetDecompressSizeAsync(
//            device_compressed_ptrs,
//            device_compressed_bytes,
//            uncompressed_bytes,
//            batch_size,
//            stream);
//    nvcompStatus_t* device_statuses;
//    hipMalloc(&device_statuses, sizeof(nvcompStatus_t)*batch_size);
//    size_t decomp_temp_bytes;
//    CHECK_NVCOMP(nvcompBatchedCascadedDecompressGetTempSize(batch_size, chunk_size, &decomp_temp_bytes), __FILE__, __LINE__);
//    void * device_decomp_temp;
//    hipMalloc(&device_decomp_temp, decomp_temp_bytes);
//    size_t* device_actual_uncompressed_bytes;
//    hipMalloc(&device_actual_uncompressed_bytes, sizeof(size_t)*batch_size);
//    CHECK_NVCOMP(nvcompBatchedCascadedDecompressAsync(device_compressed_ptrs,
//                                                      device_compressed_bytes,
//                                                      uncompressed_bytes,
//                                                      device_actual_uncompressed_bytes,
//                                                      batch_size,
//                                                      device_decomp_temp,
//                                                      decomp_temp_bytes,
//                                                      uncompressed_ptrs,
//                                                      device_statuses, stream), __FILE__, __LINE__);
    //debug start
//    size_t* host_actual_uncompressed_bytes;
//    hipHostMalloc(&host_actual_uncompressed_bytes, sizeof(size_t)*batch_size);
//    hipMemcpy(host_actual_uncompressed_bytes, device_actual_uncompressed_bytes, sizeof(size_t)*batch_size, hipMemcpyDeviceToHost);
//    for (int i = 0; i < batch_size; i++)
//    {
//        log_file << "decompressed bytes: " << *(host_actual_uncompressed_bytes + i) << std::endl;
//    }
    //debug end
    //output_decompress(batch_size, host_actual_uncompressed_bytes);
//}

void GraphicsResource::output_decompress(size_t batch_size, const size_t* host_uncompressed_bytes)
{
//    void** test;
//    hipHostMalloc(&test, sizeof(size_t) * batch_size);
//    void** host_uncompressed_ptrs;
//    hipHostMalloc(&host_uncompressed_ptrs, sizeof(size_t) * batch_size);
//    hipMemcpy(host_uncompressed_ptrs, uncompressed_ptrs, sizeof(size_t) * batch_size, hipMemcpyDeviceToHost);
//    for (int i = 0; i < batch_size; i++)
//    {
//        //log_file << host_uncompressed_bytes[i] << std::endl;
//        CHECK_ERROR(hipHostMalloc(test+i, host_uncompressed_bytes[i]), __FILE__, __LINE__);
//        CHECK_ERROR(hipMemcpyAsync(test[i], host_uncompressed_ptrs[i] , host_uncompressed_bytes[i], hipMemcpyDeviceToHost, stream), __FILE__, __LINE__);
//    }
//    hipStreamSynchronize(stream);
//    std::ofstream file;
//    file.open("debug_decompress.ppm");
//    file << "P3" << std::endl
//        << "1920 1080" << std::endl
//        << "255" << std::endl;
//    for (int i = 0; i < batch_size; i++)
//    {
//        int batch_image_size = host_uncompressed_bytes[i]/sizeof(unsigned char)/4;
//        for (int j = 0; j < batch_image_size; j++)
//        {
//            unsigned char* c = (unsigned char*)test[i] + j * 4;
//            file << (int)*c << ' '
//                 << (int)*(c + 1) << ' '
//                 << (int)*(c + 2) << std::endl;
//        }
//    }
    //hipFree(test);
//    file.close();
}

UNITY_INTERFACE_EXPORT void SendTextureIDToCuda(int texture_id, int width, int height)
{
    if (graphicsResource == nullptr)
    {
        graphicsResource = new GraphicsResource(texture_id, width, height);
    }
}

static void UNITY_INTERFACE_API OnRenderEvent(int eventID)
{
    if (!log_file.is_open())
        log_file.open("error_log.txt");
    log_file << graphicsResource->count++ << std::endl;
	graphicsResource->registerTexture();
    if (!graphicsResource->isRegistered)
        return;
	graphicsResource->mapResource();
	graphicsResource->copyCudaArray();
    graphicsResource->compress();
	graphicsResource->unmapResource();
}

UnityRenderingEvent UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API GetRenderEventFunc()
{
	return OnRenderEvent;
}

UNITY_INTERFACE_EXPORT void Dispose()
{
    graphicsResource->unmapResource();
    graphicsResource->unregisterResource();
	log_file.close();
}

