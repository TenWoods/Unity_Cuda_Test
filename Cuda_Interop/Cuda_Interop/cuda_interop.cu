#include "hip/hip_runtime.h"
﻿#include "cuda_interop.h"
#include <iostream>
#include "nvcomp/cascaded.h"

void GraphicsResource::registerTexture()
{
	CHECK_ERROR(hipGraphicsGLRegisterImage(&resource, id, GL_TEXTURE_2D, cudaGraphicsMapFlagsNone), __FILE__, __LINE__);
}

void GraphicsResource::mapResource()
{
	CHECK_ERROR(hipGraphicsMapResources(1, &resource, stream), __FILE__, __LINE__);
}

void GraphicsResource::copyCudaArray()
{
	CHECK_ERROR(hipMalloc(&data_pointer, data_length), __FILE__, __LINE__);
	CHECK_ERROR(hipGraphicsSubResourceGetMappedArray(&array, resource, 0, 0), __FILE__, __LINE__);
    CHECK_ERROR(hipMemcpy2DFromArray(data_pointer, width * sizeof(uchar4), array, 0, 0, width * sizeof(uchar4), height, hipMemcpyDeviceToDevice), __FILE__, __LINE__);
    //Debug
	//output_for_debug();
}

void GraphicsResource::unmapResource()
{
	CHECK_ERROR(hipGraphicsUnmapResources(1, &resource, stream), __FILE__, __LINE__);
    //CHECK_ERROR(hipStreamSynchronize(stream), __FILE__, __LINE__);
}

void GraphicsResource::unregisterResource()
{
	CHECK_ERROR(hipGraphicsUnregisterResource(resource), __FILE__, __LINE__);
}

void GraphicsResource::output_for_debug()
{
	if (!isFirstDebug)
		return;
	isFirstDebug = false;
	std::ofstream file;
	file.open("debug.ppm");
	void* test = malloc(data_length);
	CHECK_ERROR(hipMemcpy(test, data_pointer, data_length, hipMemcpyDeviceToHost), __FILE__, __LINE__);
	file << "P3" << std::endl
		<< "1920 1080" << std::endl
		<< "255" << std::endl;
	int texture_size = width * height;
	for (int i = 0; i < texture_size; i++)
	{
		unsigned char* c = (unsigned char*)test + i * 4;
		file << (int)*c << ' '
			<< (int)*(c + 1) << ' '
			<< (int)*(c + 2) << std::endl;
	}
	free(test);
	file.close();
}

void GraphicsResource::compress()
{
    if (!isFirstCompress)
    {
        return;
    }
    isFirstCompress = false;
    if (!log_file.is_open())
        log_file.open("error_log.txt");
    size_t* host_uncompressed_bytes;
    const size_t chunk_size = 65536;
    const size_t batch_size = (data_length + chunk_size -1) / chunk_size;
    CHECK_ERROR(hipHostMalloc(&host_uncompressed_bytes, sizeof(size_t) * batch_size), __FILE__, __LINE__);
    for (int i = 0; i < batch_size; i++)
    {
        if (i+1 < batch_size)
        {
            host_uncompressed_bytes[i] = chunk_size;
        }
        else
        {
            host_uncompressed_bytes[i] = data_length - (chunk_size*i);
        }
    }
    void ** host_uncompressed_ptrs;
    CHECK_ERROR(hipHostMalloc(&host_uncompressed_ptrs, sizeof(size_t) * batch_size), __FILE__, __LINE__);
    for (int chunk_index = 0; chunk_index < batch_size; chunk_index++)
    {
        host_uncompressed_ptrs[chunk_index] = (char*)data_pointer + chunk_index * chunk_size;
    }

    hipMalloc(&uncompressed_bytes, sizeof(size_t) * batch_size);
    hipMalloc(&uncompressed_ptrs, sizeof(size_t) * batch_size);
    hipMemcpyAsync(uncompressed_bytes, host_uncompressed_bytes, sizeof(size_t) * batch_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(uncompressed_ptrs, host_uncompressed_ptrs, sizeof(size_t) * batch_size, hipMemcpyHostToDevice, stream);

    CHECK_NVCOMP(nvcompBatchedCascadedCompressGetTempSize(batch_size, chunk_size, nvcompBatchedCascadedDefaultOpts, &temp_bytes), __FILE__, __LINE__);
    CHECK_ERROR(hipMalloc(&temp_ptr, temp_bytes), __FILE__, __LINE__);
    size_t max_out_bytes;
    CHECK_NVCOMP(nvcompBatchedCascadedCompressGetMaxOutputChunkSize(chunk_size, nvcompBatchedCascadedDefaultOpts, &max_out_bytes), __FILE__, __LINE__);
    void ** host_compressed_ptrs;
    hipHostMalloc(&host_compressed_ptrs, sizeof(size_t) * batch_size);
    for (int chunk_index = 0; chunk_index < batch_size; chunk_index++)
    {
        hipMalloc(&host_compressed_ptrs[chunk_index], max_out_bytes);
    }
    hipMalloc(&device_compressed_ptrs, sizeof(size_t) * batch_size);
    hipMemcpyAsync(device_compressed_ptrs, host_compressed_ptrs,
            sizeof(size_t) * batch_size,hipMemcpyHostToDevice, stream);
    hipMalloc(&device_compressed_bytes, sizeof(size_t) * batch_size);

    CHECK_NVCOMP(nvcompBatchedCascadedCompressAsync(uncompressed_ptrs,
                                                    uncompressed_bytes,
                                                    chunk_size, batch_size,
                                                    temp_ptr,
                                                    temp_bytes,
                                                    device_compressed_ptrs,
                                                    device_compressed_bytes,
                                                    nvcompBatchedCascadedDefaultOpts,
                                                    stream), __FILE__, __LINE__);
    //debug start
//    size_t* host_out_bytes;
//    hipHostMalloc(&host_out_bytes, sizeof(size_t)*batch_size);
//    hipMemcpy(host_out_bytes, device_compressed_bytes, sizeof(size_t) * batch_size, hipMemcpyDeviceToHost);
//    for (int i = 0; i < batch_size; i++)
//    {
//        log_file << "compressed bytes: " << *(host_out_bytes + 1) << std::endl;
//    }
    //debug end

    //decompress
    nvcompBatchedCascadedGetDecompressSizeAsync(
            device_compressed_ptrs,
            device_compressed_bytes,
            uncompressed_bytes,
            batch_size,
            stream);
    nvcompStatus_t* device_statuses;
    hipMalloc(&device_statuses, sizeof(nvcompStatus_t)*batch_size);
    size_t decomp_temp_bytes;
    CHECK_NVCOMP(nvcompBatchedCascadedDecompressGetTempSize(batch_size, chunk_size, &decomp_temp_bytes), __FILE__, __LINE__);
    void * device_decomp_temp;
    hipMalloc(&device_decomp_temp, decomp_temp_bytes);
    size_t* device_actual_uncompressed_bytes;
    hipMalloc(&device_actual_uncompressed_bytes, sizeof(size_t)*batch_size);
    CHECK_NVCOMP(nvcompBatchedCascadedDecompressAsync(device_compressed_ptrs,
                                                      device_compressed_bytes,
                                                      uncompressed_bytes,
                                                      device_actual_uncompressed_bytes,
                                                      batch_size,
                                                      device_decomp_temp,
                                                      decomp_temp_bytes,
                                                      uncompressed_ptrs,
                                                      device_statuses, stream), __FILE__, __LINE__);
    //debug start
//    size_t* host_actual_uncompressed_bytes;
//    hipHostMalloc(&host_actual_uncompressed_bytes, sizeof(size_t)*batch_size);
//    hipMemcpy(host_actual_uncompressed_bytes, device_actual_uncompressed_bytes, sizeof(size_t)*batch_size, hipMemcpyDeviceToHost);
//    for (int i = 0; i < batch_size; i++)
//    {
//        log_file << "decompressed bytes: " << *(host_actual_uncompressed_bytes + 1) << std::endl;
//    }
    //debug end
    output_decompress(chunk_size, batch_size);
}

void GraphicsResource::output_decompress(size_t chunk_size, size_t batch_size)
{
    std::ofstream file;
    file.open("debug_decompress.ppm");
    void** test;
    hipHostMalloc(&test, sizeof(size_t) * batch_size);
    //CHECK_ERROR(hipMemcpy(test, device_compressed_ptrs, sizeof(size_t) * batch_size, hipMemcpyDeviceToHost), __FILE__, __LINE__);

    file << "P3" << std::endl
    << "1920 1080" << std::endl
    << "255" << std::endl;
    hipMemcpy(test, uncompressed_ptrs , sizeof(size_t)*batch_size, hipMemcpyDeviceToHost);
//    for (int i = 0; i < batch_size; i++)
//    {
//        if ((i+1) < batch_size)
//        {
//            hipMemcpy(*(test + i), *(uncompressed_ptrs + i), chunk_size, hipMemcpyDeviceToHost);
//        }
//        else
//        {
//            hipMemcpy(*(test + i), *(uncompressed_ptrs + i), data_length - (chunk_size*i), hipMemcpyDeviceToHost);
//        }
//    }
//    int char_num = chunk_size / sizeof(unsigned char);
//    int  count = 0;
//    file << "chunk size: " << chunk_size << "batch size: " << batch_size << std::endl;
//    file << chunk_size * batch_size << std::endl;
//    file << data_length;
//    for (int i = 0; i < batch_size-1; i++)
//    {
//        for (int j = 0; j < char_num; j++)
//        {
//            file << (unsigned int)*((char *)(test + i) + j);
//            if (count < 2)
//            {
//                file << ' ';
//                count++;
//            }
//            else
//            {
//                file << std::endl;
//                count = 0;
//            }
//        }
//    }
    hipFree(test);
    file.close();
}

UNITY_INTERFACE_EXPORT void SendTextureIDToCuda(int texture_id, int width, int height)
{
    if (graphicsResource == NULL)
    {
        graphicsResource = new GraphicsResource(texture_id, width, height);
    }
}

static void UNITY_INTERFACE_API OnRenderEvent(int eventID)
{
	graphicsResource->registerTexture();
	graphicsResource->mapResource();
	graphicsResource->copyCudaArray();
    graphicsResource->compress();
	graphicsResource->unmapResource();
	graphicsResource->unregisterResource();
}

UnityRenderingEvent UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API GetRenderEventFunc()
{
	return OnRenderEvent;
}

UNITY_INTERFACE_EXPORT void Dispose()
{
	hipGraphicsUnmapResources(1, &graphicsResource->resource, 0);
    //hipStreamSynchronize(graphicsResource->stream);
	log_file.close();
}

